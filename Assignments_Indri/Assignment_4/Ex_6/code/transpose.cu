
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 8192 // Number of rows/columns of the matrix.
#define TILE_DIM 32
#define SIZE N*N // Total size of a matrix.


// Compares two matrices element by element.
int isTransposed (const double* a, const double* b, const int dim) {
  for (int i = 0; i < dim; i++) {
    for (int j = 0; j < dim; j++) {
      if(b[j + i*dim] != a[i + j*dim]) return 0;
    }
  }
  return 1;
}

// Gpu naive transposition.
__global__ void gpuNaiveTrans (const double* a, double* b, const int dim) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;
  b[row + col*dim] =  a[row*dim + col];
}

// Cpu optimised transposition.
__global__ void gpuOptTrans (const double* a, double* b, const int dim) {
  __shared__ double tmp[TILE_DIM][TILE_DIM];
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;
  tmp[threadIdx.x][threadIdx.y] = a[row*dim + col];
  __syncthreads();
  b[row + col*dim] =  tmp[threadIdx.x][threadIdx.y];
}


void matrixFill (double* a, const int dim) {
  for(int i = 0; i < dim; i++) {
    a[i] = (double) i;
  }
}


int main(int argc, char* argv[]) {


  double* hostInput, * hostOutput;
  double* devInput, * devOutput;

  // Allocate host memory.
  hostInput = (double* )malloc(SIZE * sizeof(double));
  hostOutput = (double* )malloc(SIZE * sizeof(double));

  // Allocate device memory.
  hipMalloc((void**)&devInput, SIZE * sizeof(double));
  hipMalloc((void**)&devOutput, SIZE * sizeof(double));

  // Dimension.
  dim3 grid, block;
  block.x = atoi(argv[1]);
  block.y = atoi(argv[2]);
  grid.x = N/block.x;
  grid.y = N/block.y;
  printf("\n--------------------------\n");
  printf("--------------------------\n\n");
  printf("Dimensions: block.x = %d, block.y = %d\n", block.x, block.y);


  // Initialise matricx.
  matrixFill(hostInput, SIZE);


  // Copy input to device.
  hipMemcpy(devInput, hostInput, SIZE * sizeof(double), hipMemcpyHostToDevice);

  //// Timing.
  float elapsedTime = 0.0;
  hipEvent_t tStart, tEnd;
  hipEventCreate(&tStart);
  hipEventCreate(&tEnd);


  hipEventRecord(tStart);
    gpuNaiveTrans<<< grid, block >>>(devInput, devOutput, N);
  hipEventRecord(tEnd);

  hipEventSynchronize(tEnd);
  hipEventElapsedTime(&elapsedTime, tStart, tEnd);
  printf("Elapsed time: %fms\n", elapsedTime);
  printf("Bandwidth: %f GB/s\n", 2 * SIZE * sizeof(double) / elapsedTime / 1000000);

  // Copy output to host.
  hipMemcpy(hostOutput, devOutput, SIZE * sizeof(double), hipMemcpyDeviceToHost);
  
  printf("Is the tranposition correct? %s\n",
         isTransposed(hostOutput, hostInput, N) ? "CORRECT" : "ERROR!" );


  // Freeing resources.
  free(hostInput);
  free(hostOutput);
  hipFree(devInput);
  hipFree(devOutput);
  hipEventDestroy(tStart);
  hipEventDestroy(tEnd);




}
