
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 8192 // Number of rows/columns of the matrix.
#define TILE_DIM 32
#define SIZE N*N // Total size of a matrix.


// Compares two matrices element by element.
int isTransposed (const int* a, const int* b, const int dim) {
  for (int i = 0; i < dim; i++) {
    for (int j = 0; j < dim; j++) {
      if(b[j + i*dim] != a[i + j*dim]) return 0;
    }
  }
  return 1;
}

// Gpu naive transposition.
__global__ void gpuNaiveTrans (const int* a, int* b, const int dim) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;
  b[row + col*dim] =  a[row*dim + col];
}

// Cpu optimised transposition.
__global__ void gpuOptTrans (const int* a, int* b, const int dim) {
  __shared__ double tmp[TILE_DIM][TILE_DIM];
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;
  tmp[threadIdx.x][threadIdx.y] = a[row*dim + col];
  __syncthreads();
  b[row + col*dim] =  tmp[threadIdx.x][threadIdx.y];
}


void matrixFill (int* a, const int dim) {
  for(int i = 0; i < dim; i++) {
    a[i] = i + 1;
  }
}


int main(int argc, char* argv[]) {


  int* hostInput, * hostOutput;
  int* devInput, * devOutput;

  // Allocate host memory.
  hostInput = (int* )malloc(SIZE * sizeof(int));
  hostOutput = (int* )malloc(SIZE * sizeof(int));

  // Allocate device memory.
  hipMalloc((void**)&devInput, SIZE * sizeof(int));
  hipMalloc((void**)&devOutput, SIZE * sizeof(int));

  // Dimension.
  dim3 grid, block;
  block.x = atoi(argv[1]);
  block.y = atoi(argv[2]);
  grid.x = N/block.x;
  grid.y = N/block.y;
  printf("\n--------------------------\n");
  printf("--------------------------\n\n");
  printf("Dimensions: block.x = %d, block.y = %d\n", block.x, block.y);


  // Initialise matricx.
  matrixFill(hostInput, SIZE);


  // Copy input to device.
  hipMemcpy(devInput, hostInput, SIZE * sizeof(int), hipMemcpyHostToDevice);

  //// Timing.
  float elapsedTime = 0.0;
  hipEvent_t tStart, tEnd;
  hipEventCreate(&tStart);
  hipEventCreate(&tEnd);


  hipEventRecord(tStart);
    gpuNaiveTrans<<< grid, block >>>(devInput, devOutput, N);
  hipEventRecord(tEnd);

  hipEventSynchronize(tEnd);
  hipEventElapsedTime(&elapsedTime, tStart, tEnd);
  printf("Elapsed time: %fms\n", elapsedTime);

  // Copy output to host.
  hipMemcpy(hostOutput, devOutput, SIZE * sizeof(int), hipMemcpyDeviceToHost);
  
  printf("Is the tranposition correct? %s\n",
         isTransposed(hostOutput, hostInput, N) ? "CORRECT" : "ERROR!" );


  // Freeing resources.
  free(hostInput);
  free(hostOutput);
  hipFree(devInput);
  hipFree(devOutput);
  hipEventDestroy(tStart);
  hipEventDestroy(tEnd);




}
